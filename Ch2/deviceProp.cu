#include "hip/hip_runtime.h"
#include <stdio.h>

int main(void) {
  hipDeviceProp_t deviceProp;
  int dev = 0;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("Device number %d has name %s\n", dev, deviceProp.name);
  printf("Clock freq. (KHz): %d\n", deviceProp.clockRate);
  printf("The max grid size in x: %d, y: %d, z: %d\n", deviceProp.maxGridSize[0], \
     deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);

  return 0;
}
