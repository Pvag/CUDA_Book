// Ex. 4
// =====
// Compile withouth the flag -arch sm_20
//
// The result is: it works as before (with the flag), on my platform.


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void helloFromGPU() {
  printf("Hello World from GPU!\n");
}

int main(int argc, char *argv[]) {
  // Hello from CPU
  printf("Hello World from CPU!\n");

  helloFromGPU<<<1, 10>>>();
  hipDeviceReset();

  return 0;
}
