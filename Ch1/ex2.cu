// Ex. 2
// =====
// Remove the call to cudaDeviceReset()
//
// The result is: nothing is printed from the device to the shell.


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void helloFromGPU() {
  printf("Hello World from GPU!\n");
}

int main(int argc, char *argv[]) {
  // Hello from CPU
  printf("Hello World from CPU!\n");

  helloFromGPU<<<1, 10>>>();
//  cudaDeviceReset();

  return 0;
}
