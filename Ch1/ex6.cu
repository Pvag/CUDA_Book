// Ex. 6
// =====
// Modify the kernel so that each thread will also include its number.


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void helloFromGPU() {
  printf("Hello World from thread number %d!\n", threadIdx.x);
}

int main(int argc, char *argv[]) {
  // Hello from CPU
  printf("Hello World from CPU!\n");

  helloFromGPU<<<1, 10>>>();
  hipDeviceReset();

  return 0;
}
