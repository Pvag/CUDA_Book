// Ex. 3
// =====
// Substitute cudaDeviceSynchronize() to the call to cudaDeviceReset()
//
// The result is: the device now prints to the shell, like with cudaDeviceReset().


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void helloFromGPU() {
  printf("Hello World from GPU!\n");
}

int main(int argc, char *argv[]) {
  // Hello from CPU
  printf("Hello World from CPU!\n");

  helloFromGPU<<<1, 10>>>();
  hipDeviceSynchronize();

  return 0;
}
